#include "hip/hip_runtime.h"

#include  "../common/cuda_helpers.c"

extern "C"{

#include "sum_helpers.c"

}


typedef double(*pointFunction_t)(double,double);

__device__ double max_binary_op(double a,double b){

	return fmax(a,b);

}

__device__ pointFunction_t h_binary_op = max_binary_op;


__global__ void sum_kernel(double *a, double *b ,pointFunction_t p_binary_op ,double *output,int numElements)
{
	
	const int index = threadIdx.x  + blockIdx.x * blockDim.x;

	if(index >= numElements){return;}

	output[index] = p_binary_op(a[index], b[index]);

}



int
main(void)
{
	unsigned const int num_elements = 1<<13;
	const size_t size = num_elements*sizeof(double);

	double *h_a_orig_input;
	double *h_b_orig_input;
	double *h_output;
	double *h_b_input;
	double *h_a_input;

        double *d_output;
	double *d_a_input;
	double *d_b_input;


	initialize_host(size,&h_a_input,&h_b_input,&h_output);

	  for(int i = 0; i < num_elements;i++){
                h_a_input[i] = (double)i;
		h_b_input[i] = (double)0;
                h_output[i] = 0;
        }

	initialize_device(size,&d_a_input,&d_b_input,&d_output);
	copy_host_to_device(size,h_a_input,h_b_input,h_output,d_a_input,d_b_input,d_output);
	
	h_a_orig_input = (double *)malloc(size);
	h_b_orig_input = (double *)malloc(size);

	memcpy(h_a_orig_input,h_a_input,size);
	memcpy(h_b_orig_input,h_b_input,size);
	
	pointFunction_t d_binary_op;

	hipMemcpyFromSymbol(&d_binary_op, HIP_SYMBOL(h_binary_op), sizeof(pointFunction_t));


	int threadsPerBlock = 128;
	int blocksPerGrid =(num_elements + threadsPerBlock - 1) / threadsPerBlock;
    	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    	sum_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a_input,d_b_input,d_binary_op ,d_output, num_elements);
        copy_device_to_host(size,h_a_input,h_b_input,h_output,d_a_input,d_b_input,d_output);
        check_binary_op(h_a_orig_input,h_b_orig_input,h_output,num_elements,blocksPerGrid);
	printf("DESTROYING\n");	
	destroy_host(h_a_input,h_b_input,h_output);
	destroy_device(d_a_input,d_b_input,d_output);
	free(h_a_orig_input);
	free(h_b_orig_input);
}
